#include "hip/hip_runtime.h"
/*
 *
 *	Author: Jon-Michael Hoang
 *
 *	What this is: This is a ray tracer program that I developed based off of some textbooks, papers and blogs (citations coming soon).
 *				  What ray tracing is is that it's an algorithm in computer graphics that enables
 *				  one to render life-like light and shadows.
 *
 *	How to use this: Simply compile and go to the directory of this program to find a
 *					 "raytrace_output.ppm" file, toss it into a NetPBM viewer and see the results
 *					 The reason as to why I'm doing this is because it allows for faster computation.
 *					 However, I will later find a way to use this with OpenGL.
 *
 */

#pragma once

// standard libraries used for this project
#include <iostream>
#include <time.h>
#include <fstream>

// CUDA libraries used for this project
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"
#include ""

// programmer-defined includes
#include "vec3.h"
#include "float.h"
#include "camera.h"
#include "material.h"
#include "hitable.h"
#include "hitablelist.h"
#include "ray.h"
#include "sphere.h"
#include "vec3.h"

// TODO: probably have glut work?
#include "GL\freeglut.h"

using namespace std;

// every CUDA API call that is made will return an error code that should be checked
// in this function, catch it and handle it
#define checkCudaErrors(val) checkCuda((val), #val, __FILE__, __LINE__)

void checkCuda(hipError_t result,
			   char const* const function,
			   const char* const file,
			   int const line)
{
	if (result)
	{
		cerr << "CUDA error = " << static_cast<unsigned int> (result)
			 << " at " << file << ": " << line
			 << " [" << function << "] \n";

		// make sure to call the device reset before exiting
		hipDeviceReset();

		exit(99);
	}
}



__global__ void randInit(hiprandState* randState)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
		hiprand_init(1337, 0, 0, randState);
}



/*
	TODO: Comment
*/
__device__ vec3 color(const ray& r, hitable** world, int depth, hiprandState *randState)
{
	ray currentRay = r;
	vec3 currentAttenuation = vec3(1.0, 1.0, 1.0);

	// 50 iterations
	for (int x = 0; x < 50; x++)
	{
		hitRecord rec;

		if ((*world)->hit(currentRay, 0.01, FLT_MAX, rec))
		{
			ray scattered;
			vec3 attenuation;

			if (rec.mat_ptr->scatter(currentRay, rec, attenuation, scattered, randState))
			{
				currentAttenuation *= attenuation;

				currentRay = scattered;
			}
			else
				return vec3(0.0, 0.0, 0.0);
		}
		else
		{
			vec3 unitDirection = unit_vector(currentRay.direction());

			float t = 0.5 * (unitDirection.y() + 1.0);

			// make the background with
			// linear interpolation
			vec3 c = (1.0 - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);

			return currentAttenuation * c;
		}
	}

	return vec3(0, 0, 0);
}



__global__ void renderInit(int xMax, int yMax, hiprandState* randState)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ((x >= xMax) || (y >= yMax))
		return;

	int pixelIndex = y * xMax + x;

	// threads have the same seed with different sequences and no offsets
	hiprand_init(1337, pixelIndex, 0, &randState[pixelIndex]);
}



__global__ void render(vec3* fb, int xMax, int yMax, int numSamples, camera **cam, hitable **world, hiprandState *randState)
{
	// identify the coordinates of each thread in the image so 
	// the program knows how to calculate the final color
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ((x >= xMax) || (y >= yMax))
		return;

	int pixelIndex = y * xMax + x;

	hiprandState thisRandState = randState[pixelIndex];

	vec3 col(0, 0, 0);

	for (int sample = 0; sample < numSamples; sample++)
	{
		float u = float(x + hiprand_uniform(&thisRandState)) / float(xMax);
		float v = float(y + hiprand_uniform(&thisRandState)) / float(yMax);

		ray r = (*cam)->getRay(u, v, &thisRandState);

		col += color(r, world, 0, &thisRandState);
	}

	randState[pixelIndex] = thisRandState;

	col /= float(numSamples);

	fb[pixelIndex] = vec3(sqrt(col[0]), sqrt(col[1]), sqrt(col[2]));
}



__global__ void randomScene(hitable** list, hitable** world, camera** cam, int nx, int ny, hiprandState* randState) 
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		hiprandState* thisRandState = randState;

		list[0] = new sphere(vec3(0, -1000, -1), 1000,
				  new lambertian(vec3(.5, .5, .5)));

		int x = 1;

		for (int a = -11; a < 11; a++)
		{
			for (int b = -11; b < 11; b++)
			{
				float chooseMat = hiprand_uniform(thisRandState);

				vec3 center(a + .9 * hiprand_uniform(thisRandState),
							.2,
							b + .9 * hiprand_uniform(thisRandState));

				// diffuse
				if (chooseMat < .8)
				{
					list[x++] = new movingSphere(center,
												 center + vec3(0, .5 * hiprand_uniform(thisRandState), 0),
												 0, 1, .2,
												 new lambertian(vec3(hiprand_uniform(thisRandState) * hiprand_uniform(thisRandState),
																	 hiprand_uniform(thisRandState) * hiprand_uniform(thisRandState),
																	 hiprand_uniform(thisRandState) * hiprand_uniform(thisRandState))));
				}
				// metal
				else if (chooseMat < .95)
				{
					list[x++] = new sphere(center,
										   .2,
										   new metal (vec3(.5 * (1 + hiprand_uniform(thisRandState)),
														   .5 * (1 + hiprand_uniform(thisRandState)),
														   .5 * (1 + hiprand_uniform(thisRandState)))));
				}
				// glass
				else
				{
					list[x++] = new sphere(center, .2, new dielectric(1.5));
				}
			}
		}

		list[x++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
		list[x++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
		list[x++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));

		randState = thisRandState;

		*world = new hitableList(list, 22 * 22 + 1 + 3);

		vec3 lookfrom(13, 2, 3);
		vec3 lookat(0, 0, 0);
		vec3 vup(0, 1, 0);

		float distToFocus = 10;
		float aperture = 0;

		*cam = new camera(lookfrom, 
						  lookat, 
						  vup, 
						  30, 
						  float(nx) / float(ny), 
						  aperture, 
						  distToFocus, 
						  0, 1.0);
	}
}



__global__ void freeWorld(hitable** list, hitable** world, camera** cam) 
{
	for (int x = 0; x < 404; x++) 
	{
		delete ((sphere*)list[x])->mat;
		delete list[x];
	}
	delete* world;
	delete* cam;
}



void startRayTracingProgram()
{
	// scene dimensions
	int nx = 800;
	int ny = 400;

	// number of samples per pixel
	int numSamples = 400;

	// divide the work on the GPU into tx x ty blocks of threads
	int tx = 16;
	int ty = 16;

	// as of current, the program writes to a ppm file.
	// maybe create an application window to display result?
	ofstream outfile;
	outfile.open("raytrace_cuda_output.ppm");

	// allocate numPixels sized frame buffer on the host to hold the RGB values calculated
	// by the GPU
	int numPixels = nx * ny;
	size_t fbSize = numPixels * sizeof(vec3);

	// allocate frame buffer
	vec3* frameBuffer;

	// allocate unified memory
	checkCudaErrors(hipMallocManaged((void**)& frameBuffer, fbSize));

	// allocate randomState
	hiprandState* randState;
	checkCudaErrors(hipMalloc((void**)&randState, numPixels * sizeof(hiprandState)));

	hiprandState* randState2;
	checkCudaErrors(hipMalloc((void**)&randState2, sizeof(hiprandState)));

	// 2nd random state initialized for world creation
	randInit << <1, 1 >> > (randState2);

	checkCudaErrors(hipGetLastError());
	// let the CPU know that the GPU is done
	checkCudaErrors(hipDeviceSynchronize());

	//
	// allocate scene's objects onto the GPU with hipMalloc
	//

	// creates a list of hitable objects
	hitable** list;
	int numHitables = 22 * 22 + 1 + 3;

	checkCudaErrors(hipMalloc((void**) & list, numHitables * sizeof(hitable*)));

	// creates the world that will contain the objects
	hitable** world;

	checkCudaErrors(hipMalloc((void**)& world, sizeof(hitable*)));

	// and the camera that would allow use to look at the world in a certain
	// camera angle
	camera** cam;
	
	checkCudaErrors(hipMalloc((void**)& cam, sizeof(camera*)));

	randomScene << <1, 1 >> > (list, world, cam, nx, ny, randState2);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// render the buffer
	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);

	renderInit << <blocks, threads >> > (nx, ny, randState);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	render << <blocks, threads >> > (frameBuffer, nx, ny, numSamples, cam, world, randState);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	outfile << "P3\n"
			<< nx << " "
			<< ny << " "
			<< "\n255\n";

	cerr << "P3\n"
		<< nx << " "
		<< ny << " "
		<< "\n255\n";

	// output the frame buffer into the ppm file
	for (int y = ny - 1; y >= 0; y--)
	{
		for (int x = 0; x < nx; x++)
		{
			size_t pixelIndex = y * nx + x;

			int ir = int(255.99 * frameBuffer[pixelIndex].r());
			int ig = int(255.99 * frameBuffer[pixelIndex].g());
			int ib = int(255.99 * frameBuffer[pixelIndex].b());

			outfile << ir << " "
					<< ig << " "
					<< ib << "\n";

			cerr << ir << " "
				<< ig << " "
				<< ib << "\n";
		}
	}

	// clean up
	checkCudaErrors(hipDeviceSynchronize());

	freeWorld << <1, 1 >> > (list, world, cam);

	// TODO: somehow, freeing up these causes a seg-fault;
	//       need to fix.
	checkCudaErrors(hipFree(cam));
	checkCudaErrors(hipFree(world));
	checkCudaErrors(hipFree(list));
	checkCudaErrors(hipFree(randState));
	checkCudaErrors(hipFree(randState2));
	checkCudaErrors(hipFree(frameBuffer));

	outfile.close();

	cout << "Ray-Tracing complete" << endl;

	hipDeviceReset();
}



/*

	Main

*/
int main(int argc, char** argv)
{
	std::cout << "Running the Ray-Tracing program, this might take awhile..." << endl;

	// time how long it takes for the thing to finish
	clock_t initialTime = clock();
	double duration = 0.0;

	startRayTracingProgram();

	duration = (clock() - initialTime) / (double)CLOCKS_PER_SEC;

	std::cout << "Duration: " << int(duration / 60) << " minutes and " << fmod(duration, 60) << " seconds." << endl;

	return 0;
}
/*

	END: Main

*/