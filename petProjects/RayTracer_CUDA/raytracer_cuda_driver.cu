#include "hip/hip_runtime.h"
/*
 *
 *	Author: Jon-Michael Hoang
 *
 *	What this is: This is a ray tracer program that I developed based off of some textbooks, papers and blogs (citations coming soon).
 *				  What ray tracing is is that it's an algorithm in computer graphics that enables
 *				  one to render life-like light and shadows.
 *
 *	How to use this: Simply compile and go to the directory of this program to find a
 *					 "raytrace_output.ppm" file, toss it into a NetPBM viewer and see the results
 *					 The reason as to why I'm doing this is because it allows for faster computation.
 *					 However, I will later find a way to use this with OpenGL.
 *
 */

#pragma once

// standard libraries used for this project
#include <iostream>
#include <time.h>
#include <fstream>

// CUDA libraries used for this project
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime.h"
#include ""

// programmer-defined includes
#include "vec3.h"
#include "float.h"
#include "camera.h"
#include "material.h"
#include "hitable.h"
#include "hitablelist.h"
#include "ray.h"
#include "sphere.h"
#include "vec3.h"
#include "movingSphere.h"

// TODO: probably have glut work?
#include "GL\freeglut.h"

using namespace std;

// every CUDA API call that is made will return an error code that should be checked
// in this function, catch it and handle it
#define checkCudaErrors(val) checkCuda((val), #val, __FILE__, __LINE__)

void checkCuda(hipError_t result,
			   char const* const function,
			   const char* const file,
			   int const line)
{
	if (result)
	{
		cerr << "CUDA error = " << static_cast<unsigned int> (result)
			 << " at " << file << ": " << line
			 << " [" << function << "] \n";

		// make sure to call the device reset before exiting
		hipDeviceReset();

		exit(99);
	}
}



__global__ void randInit(hiprandState* randState)
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
		hiprand_init(1337, 0, 0, randState);
}



/*
	TODO: Comment
*/
__device__ vec3 color(const ray& r, hitable** world, int depth, hiprandState *randState)
{
	ray currentRay = r;
	vec3 currentAttenuation = vec3(1.0f, 1.0f, 1.0f);

	// 50 iterations
	for (int x = 0; x < 50; x++)
	{
		hitRecord rec;

		if ((*world)->hit(currentRay, 0.01f, FLT_MAX, rec))
		{
			ray scattered;
			vec3 attenuation;

			if (rec.mat_ptr->scatter(currentRay, rec, attenuation, scattered, randState))
			{
				currentAttenuation *= attenuation;

				currentRay = scattered;
			}
			else
				return vec3(0.0f, 0.0f, 0.0f);
		}
		else
		{
			vec3 unitDirection = unit_vector(currentRay.direction());

			float t = 0.5f * (unitDirection.y() + 1.0f);

			// make the background with
			// linear interpolation
			vec3 c = (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.5f, 0.7f, 1.0f);

			return currentAttenuation * c;
		}
	}

	return vec3(0.0f, 0.0f, 0.0f);
}



__global__ void renderInit(int xMax, int yMax, hiprandState* randState)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ((x >= xMax) || (y >= yMax))
		return;

	int pixelIndex = y * xMax + x;

	// threads have the same seed with different sequences and no offsets
	hiprand_init(1337, pixelIndex, 0, &randState[pixelIndex]);
}



__global__ void render(vec3* fb, int xMax, int yMax, int numSamples, camera **cam, hitable **world, hiprandState *randState)
{
	// identify the coordinates of each thread in the image so 
	// the program knows how to calculate the final color
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ((x >= xMax) || (y >= yMax))
		return;

	int pixelIndex = y * xMax + x;

	hiprandState thisRandState = randState[pixelIndex];

	vec3 col(0.0f, 0.0f, 0.0f);

	for (int sample = 0; sample < numSamples; sample++)
	{
		float u = float(x + hiprand_uniform(&thisRandState)) / float(xMax);
		float v = float(y + hiprand_uniform(&thisRandState)) / float(yMax);

		ray r = (*cam)->getRay(u, v, &thisRandState);

		col += color(r, world, 0, &thisRandState);
	}

	randState[pixelIndex] = thisRandState;

	col /= float(numSamples);

	fb[pixelIndex] = vec3(sqrt(col[0]), sqrt(col[1]), sqrt(col[2]));
}



__global__ void randomScene(hitable** list, hitable** world, camera** cam, int nx, int ny, hiprandState* randState) 
{
	if (threadIdx.x == 0 && blockIdx.x == 0)
	{
		hiprandState* thisRandState = randState;

		theTexture* checker = new checkerTexture(new constantTexture(vec3(.0f, .0f, .0f)),
							  new constantTexture(vec3(.9f, .9f, .9f)));

		list[0] = new sphere(vec3(0.0f, -1000.0f, -1), 1000.0f,
				  new lambertian(checker));

		int x = 1;

		for (int a = -11; a < 11; a++)
		{
			for (int b = -11; b < 11; b++)
			{
				float chooseMat = hiprand_uniform(thisRandState);

				vec3 center(a + .9f * hiprand_uniform(thisRandState),
							.2f,
							b + .9f * hiprand_uniform(thisRandState));

				// diffuse
				if (chooseMat < .8f)
				{
					list[x++] = new movingSphere(center,
												 center + vec3(0.0f, .5f * hiprand_uniform(thisRandState), 0.0f),
												 0.0f, 1.0f, .2f,
												 new lambertian(vec3(
												 hiprand_uniform(thisRandState) * hiprand_uniform(thisRandState),
												 hiprand_uniform(thisRandState) * hiprand_uniform(thisRandState),
												 hiprand_uniform(thisRandState) * hiprand_uniform(thisRandState))));
				}
				// metal
				else if (chooseMat < .95f)
				{
					list[x++] = new sphere(center,
										   .2f,
										   new metal (vec3(.5f * (1.0f + hiprand_uniform(thisRandState)),
														   .5f * (1.0f + hiprand_uniform(thisRandState)),
														   .5f * (1.0f + hiprand_uniform(thisRandState)))));
				}
				// glass
				else
				{
					list[x++] = new sphere(center, .2f, new dielectric(1.5f));
				}
			}
		}

		list[x++] = new sphere(vec3(0.0f, 1.0f, 0.0f), 1.0f, new dielectric(1.5f));
		list[x++] = new sphere(vec3(-4.0f, 1.0f, 0.0f), 1.0f, new lambertian(vec3(0.4f, 0.2f, 0.1f)));
		list[x++] = new sphere(vec3(4.0f, 1.0f, 0.0f), 1.0f, new metal(vec3(0.7f, 0.6f, 0.5f), 0.0f));

		randState = thisRandState;

		*world = new hitableList(list, 22 * 22 + 1 + 3);

		vec3 lookfrom(13.0f, 2.0f, 3.0f);
		vec3 lookat(0.0f, 0.0f, 0.0f);
		vec3 vup(0.0f, 1.0f, 0.0f);

		float distToFocus = 10.0f;
		float aperture = 0.0f;

		*cam = new camera(lookfrom, 
						  lookat, 
						  vup, 
						  30.0f, 
						  float(nx) / float(ny), 
						  aperture, 
						  distToFocus, 
						  0.0f, 1.0f);
	}
}



__global__ void freeWorld(hitable** list, hitable** world, camera** cam) 
{
	for (int x = 0; x < 22 * 22 + 4; x++) 
	{
		delete ((sphere*)list[x])->mat;
		delete list[x];
	}
	delete* world;
	delete* cam;
}



__host__ void startRayTracingProgram()
{
	// scene dimensions
	int nx = 800;
	int ny = 400;

	// number of samples per pixel
	int numSamples = 400;

	// divide the work on the GPU into tx x ty blocks of threads
	int tx = 16;
	int ty = 16;

	// as of current, the program writes to a ppm file.
	// maybe create an application window to display result?
	ofstream outfile("raytrace_cuda_output.ppm");
	//outfile.open("raytrace_cuda_output.ppm");

	// allocate numPixels sized frame buffer on the host to hold the RGB values calculated
	// by the GPU
	int numPixels = nx * ny;
	size_t fbSize = numPixels * sizeof(vec3);

	// allocate frame buffer
	vec3* frameBuffer;

	// allocate unified memory
	checkCudaErrors(hipMallocManaged((void**)& frameBuffer, fbSize));

	// allocate randomState
	hiprandState* randState;
	checkCudaErrors(hipMalloc((void**)&randState, numPixels * sizeof(hiprandState)));

	hiprandState* randState2;
	checkCudaErrors(hipMalloc((void**)&randState2, sizeof(hiprandState)));

	// 2nd random state initialized for world creation
	randInit << <1, 1 >> > (randState2);

	checkCudaErrors(hipGetLastError());
	// let the CPU know that the GPU is done
	checkCudaErrors(hipDeviceSynchronize());

	//
	// allocate scene's objects onto the GPU with hipMalloc
	//

	// creates a list of hitable objects
	hitable** list;
	int numHitables = 22 * 22 + 1 + 3;

	checkCudaErrors(hipMalloc((void**) & list, numHitables * sizeof(hitable*)));

	// creates the world that will contain the objects
	hitable** world;

	checkCudaErrors(hipMalloc((void**)& world, sizeof(hitable*)));

	// and the camera that would allow use to look at the world in a certain
	// camera angle
	camera** cam;
	
	checkCudaErrors(hipMalloc((void**)& cam, sizeof(camera*)));

	randomScene << <1, 1 >> > (list, world, cam, nx, ny, randState2);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	// render the buffer
	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);

	renderInit << <blocks, threads >> > (nx, ny, randState);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	render << <blocks, threads >> > (frameBuffer, nx, ny, numSamples, cam, world, randState);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	outfile << "P3\n"
			<< nx << " "
			<< ny << " "
			<< "\n255\n";

	cerr << "P3\n"
		<< nx << " "
		<< ny << " "
		<< "\n255\n";

	// output the frame buffer into the ppm file
	for (int y = ny - 1; y >= 0; y--)
	{
		for (int x = 0; x < nx; x++)
		{
			size_t pixelIndex = y * nx + x;

			int ir = int(255.99f * frameBuffer[pixelIndex].r());
			int ig = int(255.99f * frameBuffer[pixelIndex].g());
			int ib = int(255.99f * frameBuffer[pixelIndex].b());

			outfile << ir << " "
					<< ig << " "
					<< ib << "\n";

			cerr << ir << " "
				<< ig << " "
				<< ib << "\n";
		}
	}

	// clean up
	checkCudaErrors(hipDeviceSynchronize());

	//freeWorld << <1, 1 >> > (list, world, cam);

	// TODO: somehow, freeing up these causes a seg-fault;
	//       need to fix.

	try
	{
		checkCudaErrors(hipFree(cam));
		checkCudaErrors(hipFree(world));
		checkCudaErrors(hipFree(list));
		checkCudaErrors(hipFree(randState));
		checkCudaErrors(hipFree(randState2));
		checkCudaErrors(hipFree(frameBuffer));
	}
	catch (exception e)
	{
		cout << endl << e.what() << endl;
	}

	outfile.close();

	cout << "Ray-Tracing complete" << endl;

	hipDeviceReset();
}



/*

	Main

*/
int main(int argc, char** argv)
{
	std::cout << "Running the Ray-Tracing program, this might take awhile..." << endl;

	// time how long it takes for the thing to finish
	clock_t initialTime = clock();
	double duration = 0.0;

	startRayTracingProgram();

	duration = (clock() - initialTime) / (double)CLOCKS_PER_SEC;

	std::cout << "Duration: " << int(duration / 60) << " minutes and " << fmod(duration, 60) << " seconds." << endl;

	//system("xdg-open raytrace_cuda_output.ppm");

	return 0;
}
/*

	END: Main

*/